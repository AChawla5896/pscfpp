/*
* PSCF - Polymer Self-Consistent Field Theory
*
* Copyright 2016, The Regents of the University of Minnesota
* Distributed under the terms of the GNU General Public License.
*/

#include "WaveList.h"

namespace Pscf { 
namespace Pssp_gpu 
{ 

   using namespace Util;

   template class WaveList<1>;
   template class WaveList<2>;
   template class WaveList<3>;

}
}
